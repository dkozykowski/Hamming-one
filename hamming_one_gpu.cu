// Hamming One
// Architecture: GPU
// Complexity: L * M^2
//


#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cstring>
using namespace std;

#define BITS_IN_INT 31
#define ERR(source) (fprintf(stderr,"%s:%d\n",__FILE__,__LINE__),\
                     perror(source),\
                     exit(EXIT_FAILURE))

void usage(char *name){
    fprintf(stderr,"USAGE: %s <input_file_path>\n",name);
    exit(EXIT_FAILURE);
}

int _ceil(double variable) {
    int new_variable = (int)variable;
    if ((double)new_variable == variable) return new_variable;
    else return new_variable + 1;
}

void read_input(char* file_path, int& L, int& M, int*& h_input) {
    ifstream fileStream;    
    fileStream.open(file_path, ios::in);
    if (!fileStream.is_open()) ERR("ifstream.open");
    fileStream >> L >> M;

    int newL = _ceil((double)L / BITS_IN_INT);

    h_input = new int[newL * M];
    if (h_input == NULL) ERR("operator new");
    memset(h_input, 0, sizeof(int) * newL * M);

    int current_bit;
    for (int i = 0; i < M; i++) {
        for (int o = 0; o < newL; o++) {
            for (int j = 0; j < BITS_IN_INT && o * BITS_IN_INT + j < L; j++) {
                fileStream >> current_bit;
                h_input[o + i * newL] = (h_input[o + i * newL] << 1) + current_bit;
            }
        }
    }
    L = newL;
}

__global__ void find_hamming_one(int* d_input, bool* d_output, int L, int M) {
    int index = blockIdx.x * 1024 + threadIdx.x;
    int hamming_distance;
    for (int i = index + 1; i < M; i++) {
        hamming_distance = 0;

        for (int o = 0; o < L; o++) {
            int num = d_input[o + index * L] ^ d_input[o + i * L];
            while(num != 0) {
                hamming_distance += (num & 1);
                num >>= 1;
            }
        }
        d_output[i + index * M] = (hamming_distance == 1);
    }
}

int main(int argc, char ** argv) {
    if (argc != 2) usage(argv[0]);
    int L, M;
    int *h_input, *d_input;
    bool *h_output, *d_output;
    read_input(argv[1], L, M, h_input);

    hipMalloc(&d_input, L * M * sizeof(int));
    hipMemcpy(d_input, h_input, L * M * sizeof(int), hipMemcpyHostToDevice);
    delete[] h_input;

    hipMalloc(&d_output, M * M);
    hipMemset(d_output, 0, M * M);
    h_output = new bool[M * M];
    if (h_output == NULL) ERR("operator new");

    int threads, blocks;
    threads = 1024;
    blocks = _ceil((double)M / threads);

    find_hamming_one<<<blocks, threads>>>(d_input, d_output, L, M);
    hipMemcpy(h_output, d_output, M * M, hipMemcpyDeviceToHost);

    for (int i = 0; i < M; i++) {
        for (int o = i + 1; o < M; o++) {
            if (h_output[o + i * M]) {
                cout << i << " " << o << "\n";
            }
        }
    }

    hipFree(d_input);
    hipFree(d_output);
    delete[] h_output;
    return EXIT_SUCCESS;
}