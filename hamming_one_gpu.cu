
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cstring>
using namespace std;

#define BITS_IN_INT 31

#define INPUT_FILE_NAME "input.txt"

int _ceil(double variable) {
    int new_variable = (int)variable;
    if ((double)new_variable == variable) return new_variable;
    else return new_variable + 1;
}

void read_input(int& L, int& M, int*& h_input) {
    ifstream fileStream;    
    fileStream.open(INPUT_FILE_NAME, ios::in);
    if (!fileStream.is_open()) exit(-1);
    fileStream >> L >> M;

    int newL = _ceil((double)L / BITS_IN_INT);

    h_input = new int[newL * M];
    if (h_input == NULL) exit(-1);
    memset(h_input, 0, sizeof(int) * newL * M);

    int current_bit;
    for (int i = 0; i < M; i++) {
        for (int o = 0; o < newL; o++) {
            for (int j = 0; j < BITS_IN_INT && o * BITS_IN_INT + j < L; j++) {
                fileStream >> current_bit;
                h_input[o + i * newL] = (h_input[o + i * newL] << 1) + current_bit;
            }
        }
    }
    L = newL;
}

__global__ void find_hamming_one(int* d_input, bool* d_output, int L, int M) {
    int index = blockIdx.x * 1024 + threadIdx.x;
    int hamming_distance;
    for (int i = index + 1; i < M; i++) {
        hamming_distance = 0;

        for (int o = 0; o < L; o++) {
            int num = d_input[o + index * L] ^ d_input[o + i * L];
            while(num != 0) {
                hamming_distance += (num & 1);
                num >>= 1;
            }
        }
        d_output[i + index * M] = (hamming_distance == 1);
    }
}

int main(int argc, char ** argv) {
    int L, M;
    int *h_input, *d_input;
    bool *h_output, *d_output;
    read_input(L, M, h_input);

    hipMalloc(&d_input, L * M * sizeof(int));
    hipMemcpy(d_input, h_input, L * M * sizeof(int), hipMemcpyHostToDevice);
    delete[] h_input;

    hipMalloc(&d_output, M * M);
    hipMemset(d_output, 0, M * M);
    h_output = new bool[M * M];
    if (h_output == NULL) return -1;

    int threads, blocks;
    threads = 1024;
    blocks = _ceil((double)M / threads);

    find_hamming_one<<<blocks, threads>>>(d_input, d_output, L, M);
    hipMemcpy(h_output, d_output, M * M, hipMemcpyDeviceToHost);

    for (int i = 0; i < M; i++) {
        for (int o = i + 1; o < M; o++) {
            if (h_output[o + i * M]) {
                cout << i << " " << o << "\n";
            }
        }
    }

    hipFree(d_input);
    hipFree(d_output);
    delete[] h_output;
    return 0;
}